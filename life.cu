#include "hip/hip_runtime.h"
#include <stdint.h>

#include "cutil.h"
#include "life.h"

__global__ void life(uint8_t *grid, uint8_t *output_grid, int width,
      int height);

extern "C" void natural_select(uint8_t *top, uint8_t *bot, uint8_t *grid,
      int width, int height) {
   uint8_t *dev_input, *dev_output;

   CUDA_SAFE_CALL(hipMalloc((void **)&dev_input,
            ((2 * width) + (width * height)) * sizeof(uint8_t)));
   CUDA_SAFE_CALL(hipMalloc((void **)&dev_output,
            ((2 * width) + (width * height)) * sizeof(uint8_t)));

   CUDA_SAFE_CALL(hipMemcpy(dev_input, top, width * sizeof(uint8_t), TO_DEV));
   CUDA_SAFE_CALL(hipMemcpy(dev_input + width, grid,
            width * height * sizeof(uint8_t), TO_DEV));
   CUDA_SAFE_CALL(hipMemcpy(dev_input + (width * height) + width, bot,
            width * sizeof(uint8_t), TO_DEV));

   life<<<MAX_BLOCKS, THREADS_PER_BLOCK>>>(dev_input, dev_output, width,
            height);

   CUDA_SAFE_CALL(hipMemcpy(grid, dev_output + width,
            width * height * sizeof(uint8_t), TO_HOST));

   hipFree(dev_input);
   hipFree(dev_output);
}

__global__ void life(uint8_t *grid, uint8_t *output_grid, int width,
      int height) {
   // Find the focus cell, ignore the first row
   int cell_id = width + blockIdx.x * blockDim.x + threadIdx.x;
   uint8_t live_neighbors;
   int row_idx, col_idx, neighbor_id;

   while (cell_id < (width * height) + width) {

      // Iterate through neighboring blocks and count those which are alive
      live_neighbors = 0;
      for (row_idx = 0; row_idx < 3; row_idx++) {
         for (col_idx = 0; col_idx < 3; col_idx++) {

            /*
             * Find the current neighbor. Ignore if it is out of bounds or
             * the focus cell.
             */
            neighbor_id = (cell_id + col_idx - 1) + (width * (row_idx - 1));
            if (neighbor_id < 0 || neighbor_id == cell_id
                  || neighbor_id >= (width * height) + (2 * width)) {
               continue;
            }

            // Increment the count of living neighbors
            live_neighbors += grid[neighbor_id];
         }
      }

      // If 2 or 3 neighbors are alive, the focus cell lives, else, it dies
      output_grid[cell_id] = (live_neighbors == 2 || live_neighbors == 3) ? 1 : 0;

      // Move to the next focus cell.
      cell_id += blockDim.x + gridDim.x;
   }
}
